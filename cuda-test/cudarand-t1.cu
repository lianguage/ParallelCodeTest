#include "hip/hip_runtime.h"
#include <iostream>
//#include "hip/hip_runtime.h"
//#include ""
#include <hiprand/hiprand_kernel.h>
#include <ctime>
#include <cstdio>


__global__ void d_cudaRand(double *d_out)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState state;
    hiprand_init((unsigned long long)clock() + i, 0, 0, &state);

    d_out[i] = hiprand_uniform_double(&state);
}

__global__ void f_cudaRand(float *d_out){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState state;
    hiprand_init((unsigned long long)clock() + i, 0, 0, &state);

    d_out[i] = hiprand_normal(&state);

}

int main(int argc, char** argv)
{
    size_t N = 1 << 4;
    double *h_v = new double[N];

    double *d_out;
    hipMalloc((void**)&d_out, N * sizeof(double));

    // generate random numbers
    d_cudaRand << < 1, N >> > (d_out);

    hipMemcpy(h_v, d_out, N * sizeof(double), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < N; i++){
        printf("out: %f\n", h_v[i]);
    }

    hipFree(d_out);
    delete[] h_v;

    return 0;
}