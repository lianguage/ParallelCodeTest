#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

      1) Histogram of the number of occurrences o feach digit
      2) Exclusive Prefix Sum of Histogram
      3) Determine relative offset of each digit
           For example [0 0 1 1 0 0 1]
                   ->  [0 1 0 1 2 3 2]
      4) Combine the results of steps 2 &3  to determine the final
         output location for each element and move it there

      LSB Radix sort is an out-of-place sort and you will need to ping-pong values
      between the input and output buffers we have provided.  Make sure the final
      sorted results end up in the output buffer!  Hint: You may need to do a copy
      at the end.

    */
#include "assert.h"

//are these needed?
#include <hip/hip_runtime.h>
#include <iostream>
//#include <hip/hip_runtime_api.h>

#include <fstream>
#include <string>
#include <climits>

//@max_reduce tested.
__global__ void max_reduce(   unsigned int* const d_position,
                              unsigned int* const d_result,
                              int size
                           ){
   	extern __shared__ unsigned int shared[];
   	int index = threadIdx.x + blockDim.x * blockIdx.x;
   	int tid = threadIdx.x;
   	if( index < size ){
		shared[tid] = d_position[index];
	}
   	__syncthreads();


  	for( int s = blockDim.x/2 ; s > 0 ; s >>= 1 ){
    	if( tid < s){
			shared[tid] = max( shared[tid], shared[tid+s]);
    	}
      __syncthreads();
   	}

   	if(tid == 0){
      	d_result[ blockIdx.x ] = shared[tid];
   	}
}


//@radix_predicate tested
__global__ void radix_predicate(   unsigned int * const d_input,
                                   //unsigned int* const d_position,
                                   unsigned int * const d_predicate,
                                   int current_bit,
                                   int size
                                   ){
   
	int index = threadIdx.x + blockDim.x*blockIdx.x;
	if(index >= size ){ return; }

	//int position = d_position[index];
	//predicate: (i & 1) == 0;
	unsigned int x = d_input[index];
	x >>= current_bit;
	d_predicate[index] = x&1;
}


__global__ void scan_inplace_threads(
									unsigned int * const d_elements,
									int numElems
									){
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;
	int bos = blockDim.x * blockIdx.x;

	for( int s = 1 ; s < blockDim.x ; s <<= 1 ){
		unsigned int val = 0;
		int spot = tid - s;
		if( spot >= 0 && index < numElems){
	 		val = d_elements[spot+bos];
		}
		__syncthreads();
		if( spot >= 0 && index < numElems){
			d_elements[index] += val;
		}
		__syncthreads();
	}
}

__global__ void scan_get_block_sum( 
								unsigned int * const d_scanned_elements,//expected to be an inclusive scan
								unsigned int * const d_blocksums,
								int numElems
							 ){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	//if(index >= numElems ){ return; } //syncthreads not used, so early exit is not a problem
	
	if( threadIdx.x + 1 == blockDim.x && index < numElems || index+1 == numElems ){ 
		d_blocksums[blockIdx.x] = d_scanned_elements[index];
	}
}


__global__ void scan_add_block_sum(
										unsigned int * const d_elements,
										unsigned int * const d_blocksums,
										int numElems
									 ){
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	//if( index >= numElems ){ return; }
	//if(blockIdx.x != 0){d_elements[index] += d_blocksums[blockIdx.x + 1]; }
	if( index < numElems && index >= blockDim.x ){
		d_elements[index] += d_blocksums[blockIdx.x - 1];
	} 

}

__global__ void scan_polishing( 
										unsigned int * const d_elems,
										unsigned int * const d_predicate,
										unsigned int * const d_middle,
										int numElems
									  ){
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if( index >= numElems ){ return; }

	//inclusive to exclusive
	int temp = 0;
	if( index != 0 ){temp = d_elems[index -1];}
	d_elems[index] = temp;

	//adding 'identity' to all elements 
	d_elems[index] += d_middle[0];
	if( index == numElems - 1 ){ 
		d_middle[0] = d_elems[index];
		d_middle[0] += d_predicate[index];
	}

}


__global__ void radix_invert_predicate( unsigned int * const d_predicate,
                                        int size){
	int index = threadIdx.x + blockDim.x*blockIdx.x;
	if( index >= size ){ return;}
	d_predicate[index] == 0 ? d_predicate[index] = 1 : d_predicate[index] = 0;
}


__global__ void radix_reposition( 
							 unsigned int* const d_position1,
						     unsigned int* const d_position2,
                             unsigned int* const d_predicate,
                             unsigned int* const d_values,
                             int size){

	int index = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int displacement_index= 0;
	unsigned int value = 0;
	if( index < size ){
		if(d_predicate[index] == 1){
			displacement_index = d_position2[index];
		}
		else{
			displacement_index = d_position1[index];
		}
		value = d_values[index];
	}
	syncthreads();
	if(index < size && displacement_index < size ){ 
		d_values[displacement_index] = value;
	}

}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
   }
}

#define MAX_BLOCKSZ 1024

using namespace std;

int getBlockSize(int numElems){
	if(numElems > MAX_BLOCKSZ){
		return MAX_BLOCKSZ;
	}
	else{
		return numElems;
	}
}

int getGridSize(int numElems){
	return (numElems + MAX_BLOCKSZ -1)/MAX_BLOCKSZ;
}



//@find_max_reduce tested
int find_max_reduce( unsigned int* const d_position , int numElems ){
	int blockSize = getBlockSize(numElems);
	int gridSize = getGridSize(numElems);
	int outFElems = gridSize * sizeof(unsigned int); //filesize of the output from one iteration of kernel

	unsigned int * d_result; //pointer to where result of max reduce kernel can be stored
	hipMalloc( (void**)&d_result, outFElems ); 
	unsigned int * d_incopy; //a copy of input
	hipMalloc( (void**)&d_incopy, sizeof(unsigned int)*numElems );
	hipMemcpy( d_incopy, d_position, sizeof(unsigned int)*numElems, hipMemcpyDeviceToDevice );

	int shareFSize = blockSize * sizeof(unsigned int);
	
	int s = numElems; 
	do{
		int current_gridSz = getGridSize(s);
		max_reduce<<< current_gridSz, blockSize, shareFSize>>>( d_incopy, d_result, s );
		hipDeviceSynchronize();
		hipMemcpy( d_incopy, d_result, current_gridSz*sizeof(unsigned int), hipMemcpyDeviceToDevice);

		std::cout << "max_reduce called" << std::endl; //@test

		s /= blockSize;
	}while( s > blockSize );
	unsigned int result = 0;
	hipMemcpy( &result, d_result , sizeof(unsigned int), hipMemcpyDeviceToHost );
	return result;
}

#include <queue>

void scan_arbitrary( 
						unsigned int * const d_elements,
						unsigned int * const d_predicate,
						unsigned int * const d_middle,
						int numElems ){

	cout << "====arbitrary scan====" << endl;
	int gridSize = getGridSize(numElems);
	int blockSize = getBlockSize(numElems);

	//int thisGridSize = gridSize;
	//int thisNumElems = numElems;

	//deque<unsigned int*> D_scan_targets;
	//deque<int> blocksum_numelems;
	//D_scan_targets.push_back(d_elements);
	//blocksum_numelems.push_back(gridSize);

	int cycles = 0;
	for( int tempGridSize = gridSize, 
			 tempNumElems = numElems 
		    ;tempNumElems > 1
		    ;tempNumElems = tempGridSize, 
		 	 tempGridSize = getGridSize(tempNumElems),
			 cycles++
	);//After the for loop cycles ends up being the correct size
	//running this loop for i < cycles will loop the corresponding number of cycles in this loop.

	vector<unsigned int*> D_scan_targets(cycles+1);
	vector<int> blocksum_numelems(cycles+1);
	scan_inplace_threads<<<gridSize,blockSize>>>( d_elements, numElems );
	D_scan_targets[0] = d_elements;
	blocksum_numelems[0] = numElems; 

	for( int i = 0, 
		     tempGridSize = gridSize,
		     tempNumElems = numElems
		    ;i < cycles
			;i++
	){
		unsigned int * d_blockscan;
		gpuErrchk( hipMalloc( (void**)&d_blockscan , gridSize*sizeof(unsigned int) ) );
		scan_get_block_sum<<<tempGridSize,blockSize>>>(D_scan_targets[i], d_blockscan, tempNumElems);
		
		//resize dimensions to calculate blocksums
		tempNumElems = tempGridSize ;
		tempGridSize = getGridSize(tempNumElems) ;
		if( tempNumElems > 1 ){	
			scan_inplace_threads<<<tempGridSize,blockSize>>>( d_blockscan, tempNumElems );
			D_scan_targets[i+1] = d_blockscan;
			blocksum_numelems[i+1] = tempNumElems;
			
			//test
			//cout << "i = " << i + 1 << endl;
			//cout << "tempNumElems:" << tempNumElems << endl;

		}

	}

	for(int i =  cycles - 1 ; i > 0 ; i-- ){
		int tempGridSize = getGridSize( blocksum_numelems[i-1] );
		scan_add_block_sum<<<tempGridSize,blockSize>>>( D_scan_targets[i-1], D_scan_targets[i], blocksum_numelems[i-1] );
		//Consider freeing memory here
	}

	//test print loop
	for( int i = cycles-1 ; i > 0 ; i--){
		int memsize = blocksum_numelems[i] * sizeof(unsigned int);
		unsigned int * h_blockscan = (unsigned int*)malloc( memsize ) ;
		gpuErrchk( hipMemcpy( h_blockscan, D_scan_targets[i] , memsize, hipMemcpyDeviceToHost ));
		cout << "h_blockscan:"; 
		for( int j = 0 ; j < blocksum_numelems[i] ; j++ ){
			cout << h_blockscan[j] << ",";
		}
		cout << endl;
	}
	//end test print loop

	for( int i = 1 ; i < cycles ; i++ ){
		hipFree(D_scan_targets[i]);// free all the elements except the first since it is used again after function end.
	}

	//test
	/*
	for( int i = 0 ; i < cycles ; i++ ){
		cout << "blocksum_numelems#" << i << blocksum_numelems[i];
	}*/



	//int cyclestest = 0;

	/*do{
		cout << "thisGridSize:" << thisGridSize << endl;
		cout << "thisNumElems:" << thisNumElems << endl;
		//kernel operations
		scan_inplace_threads<<<thisGridSize,blockSize>>>( d_elements, numElems );
			hipDeviceSynchronize();
			//cout<< "inplace_threads" << endl;
		
		unsigned int * d_blocksums;
		gpuErrchk(hipMalloc((void**)&d_blocksums, thisGridSize*sizeof(unsigned int) ));
			hipDeviceSynchronize();
		scan_get_block_sum<<<thisGridSize,blockSize>>>( d_elements, d_blocksums, numElems );
			hipDeviceSynchronize();

		D_scan_targets.push_back( d_blocksums );
		blocksum_numelems.push_back(thisGridSize);
			//cout<< " scan_polishing " << endl;
		

		//concluding counter modification operations
		thisNumElems = thisGridSize;
		thisGridSize = getGridSize(thisNumElems);
		cyclestest++;

	}while( thisNumElems > 1 );*/
	
	//int i = 0;
	//int j = 0;
	//for(; i < cycles; i++){j++;}  //for rand1:
	//cout << "j-test:" << j << endl; //j = 4
	//cout << "i-test:" << i << endl; //i = 4
	//cout << "cycles:" << cycles << endl; //cycles = 3
	//cout << "cyctest:" << cyclestest << endl; //cyctest = 3


	scan_polishing<<<gridSize,blockSize>>>( d_elements, d_predicate, d_middle, numElems);
			hipDeviceSynchronize();
	

}

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_numbers,
               unsigned int* const d_position1,
               const size_t numElems)
{

	const int filesize = numElems*sizeof(unsigned int);
	//unsigned int* d_positionCopy;
	//unsigned int* d_inPosCopy;
	unsigned int * d_predicate;
	unsigned int* d_position2;

	gpuErrchk(hipMalloc((void**)&d_predicate, filesize ));
	gpuErrchk(hipMalloc((void**)&d_position2, filesize ));

	//set scan destination to 0;
	gpuErrchk(hipMemset( d_predicate, filesize, 0 ));
	gpuErrchk(hipMemcpy ( d_position1, d_inputPos, filesize,hipMemcpyDeviceToDevice ));
	gpuErrchk(hipMemcpy ( d_position2, d_inputPos, filesize, hipMemcpyDeviceToDevice ));
	gpuErrchk(hipMemcpy ( d_numbers, d_inputVals, filesize, hipMemcpyDeviceToDevice ));//Values changed after calculating most significant element.

	int blockSize =  getBlockSize(numElems);
	int gridSize = getGridSize(numElems);
	unsigned int * d_middle;
	gpuErrchk(hipMalloc((void**)&d_middle, sizeof(unsigned int) ));
	unsigned int bitsig = find_max_reduce( d_numbers, numElems);//find the largest element, so the number of bits to run radix is known.
	cout<< "bitsig:" << bitsig << endl;
	if( bitsig <= UINT_MAX/2 ){ bitsig <<= 1; }
	else{ bitsig = UINT_MAX; }

	//cout<< "bitsig:" << bitsig << endl;
	//cout<< "max_uint:" << UINT_MAX << endl;

	unsigned int * d_auxarray;//used by the scan kernel for storing block sums, since threads cannot be relied on being in sync.
	gpuErrchk( hipMalloc((void**)&d_auxarray, gridSize*sizeof(unsigned int) )); //needs to hold as many elements as there are blocks.

	for ( int current_bit = 0 ; (bitsig >> current_bit) > 1 ; current_bit++  ){
		//unsigned int h_middle = 0;
		cout << "currentbit:" << current_bit << endl;
		gpuErrchk(hipMemset(d_middle,0, sizeof(unsigned int)));
			hipDeviceSynchronize();

		radix_predicate<<<gridSize,blockSize>>>(d_numbers, d_predicate, current_bit, numElems);
			hipDeviceSynchronize();
			//cout<< "radix_predicate" << endl;
		radix_invert_predicate<<<gridSize,blockSize>>>(d_predicate, numElems);
			hipDeviceSynchronize();
			//cout<< "invert_predicate" << endl;
		hipMemcpy(d_position1, d_predicate, filesize, hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();

		scan_arbitrary(d_position1 ,d_predicate, d_middle, numElems);
			hipDeviceSynchronize();
			//cout<< "scan_arbitrary" << endl;
		radix_invert_predicate<<<gridSize,blockSize>>>(d_predicate, numElems);
			hipDeviceSynchronize();

		hipMemcpy(d_position2, d_predicate, filesize, hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();

		scan_arbitrary(d_position2 , d_predicate, d_middle, numElems);
			hipDeviceSynchronize();

		radix_reposition<<<gridSize,blockSize>>>(d_position1, d_position2, d_predicate, d_numbers, numElems );
			hipDeviceSynchronize();
			//cout<< "reposition" << endl;
		//cout << "current_bit: " << current_bit <<endl;
		//cout << "bitsig>>currentbit" << (bitsig >> current_bit) << endl;
		//start @test
		/*
		unsigned int * h_numbers = (unsigned int*) malloc( filesize);
		unsigned int * h_position1 = (unsigned int*) malloc(filesize);
		unsigned int * h_position2 = (unsigned int*) malloc( filesize);
		unsigned int * h_bools = (unsigned int *)malloc(filesize);
		
		gpuErrchk( hipMemcpy( h_numbers, d_numbers, filesize, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		gpuErrchk( hipMemcpy( h_bools, d_predicate, filesize, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		gpuErrchk( hipMemcpy( h_position1, d_position1, filesize, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		gpuErrchk( hipMemcpy( h_position2, d_position2, filesize, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		//hipMemcpy( h_position2, d_position2, filesize, hipMemcpyDeviceToHost);
		//hipMemcpy( h_position1, d_position1, filesize, hipMemcpyDeviceToHost);
		//hipMemcpy( h_bools, d_predicate, filesize, hipMemcpyDeviceToHost);
		//hipMemcpy( h_numbers, d_numbers, filesize, hipMemcpyDeviceToHost);

		cout << " bools : {";
		for(int i = 0 ; i < numElems ; i++){
			cout << std::to_string(h_bools[i]) << ", ";
			//std::cout << "adsf" <<std::endl; //@test
		}
		cout << "} \n";
		cout << " numbers : {";
		for(int i = 0 ; i < numElems ; i++){
			cout << std::to_string(h_numbers[i]) << ", ";
			//std::cout << "adsf" <<std::endl; //@test
		}
		cout << "} \n";
		cout << " position1 : {";
		for(int i = 0 ; i < numElems ; i++){
			cout << std::to_string(h_position1[i]) << ", ";
			//std::cout << "adsf" <<std::endl; //@test
		}
		cout << "} \n";
		cout << " position 2 : {";
		for(int i = 0 ; i < numElems ; i++){
			cout << std::to_string(h_position2[i]) << ", ";
			//std::cout << "adsf" <<std::endl; //@test
		}
		cout << "} \n";
		//end @test
		free(h_numbers);
		free(h_bools);
		free(h_position1);
		free(h_position2);
		*/
	}
	//cout << "rwar";

	//d_numbers and d_position1 are needed to be persistent after the function concludes
	//hipFree(d_numbers);
	//hipFree(d_position1);
	hipFree(d_predicate);
	hipFree(d_position2);
	hipFree(d_middle);
}

/*
int main(int argc, char * argv[]){
	if(argc !=  2){
		cout <<   "usage: " << argv[0] << "<filename>" << endl;
	}
	else{
		string line;
		ifstream myfile(argv[1]);
		int size = 0;
		int filesize = 0;
		int lines = 0;
		unsigned int * h_numbers;
		unsigned int * h_position;
		if( myfile.is_open()){
			getline(myfile,line);
			size = atoi(line.c_str());//first line of file is assumed to show number of elements in file.
			filesize = sizeof(int)*size;
			h_numbers = (unsigned int *)malloc(filesize);
			h_position = (unsigned int *)malloc(filesize);
			int i = 0;
			//cout << "moo:" << size << endl;
			while(getline(myfile, line)){
				 //cout << line << endl;
				 h_numbers[i] = atoi(line.c_str());
				 h_position[i] = i;
				 lines++;
				 i++;
			}
			myfile.close();
		}
		else {
			cout << "Sorry mate, can't load file" << endl;
			return 0;
		}
			//allocate memory for device
		unsigned int * d_numbers;
		unsigned int * d_position;
		unsigned int * d_out_numbers;
		unsigned int * d_out_position;

		gpuErrchk( hipMalloc((void**)&d_numbers, filesize));
		hipDeviceSynchronize();
		gpuErrchk( hipMalloc((void**)&d_position, filesize));
		hipDeviceSynchronize();
		gpuErrchk( hipMalloc((void**)&d_out_numbers, filesize));
		hipDeviceSynchronize();
		gpuErrchk( hipMalloc((void**)&d_out_position, filesize));
		hipDeviceSynchronize();

		//copy memory host to device
		gpuErrchk( hipMemcpy( d_numbers, h_numbers, filesize , hipMemcpyHostToDevice ));
		hipDeviceSynchronize();
		gpuErrchk( hipMemcpy( d_position, h_position, filesize, hipMemcpyHostToDevice ));
		hipDeviceSynchronize();

		//run sort
		your_sort(d_numbers, d_position, d_out_numbers, d_out_position, lines );
		//hipDeviceSynchronize();

		gpuErrchk( hipMemcpy( h_numbers, d_out_numbers ,filesize, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		ofstream sortedfile("sorted");
		if( sortedfile.is_open()){
			for(int i = 0 ; i < lines ; i++){
				sortedfile << std::to_string(h_numbers[i]) << "\n";
			}
		}
		std::cout << "finished" <<std::endl;

		return 0;

	}

}
*/