#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

      1) Histogram of the number of occurrences o feach digit
      2) Exclusive Prefix Sum of Histogram
      3) Determine relative offset of each digit
           For example [0 0 1 1 0 0 1]
                   ->  [0 1 0 1 2 3 2]
      4) Combine the results of steps 2 &3  to determine the final
         output location for each element and move it there

      LSB Radix sort is an out-of-place sort and you will need to ping-pong values
      between the input and output buffers we have provided.  Make sure the final
      sorted results end up in the output buffer!  Hint: You may need to do a copy
      at the end.

    */
#include "assert.h"

#include <hip/hip_runtime.h> //used for assert
#include <iostream>

#include <fstream>
#include <string>
#include <climits>


__global__ void max_reduce(   unsigned int* const d_position,
                              unsigned int* const d_result,
                              int size
                           ){
   	extern __shared__ unsigned int shared[];
   	int index = threadIdx.x + blockDim.x * blockIdx.x;
   	int tid = threadIdx.x;
   	if( index < size ){
		shared[tid] = d_position[index];
	}
   	__syncthreads();


  	for( int s = blockDim.x/2 ; s > 0 ; s >>= 1 ){
    	if( tid < s){
			shared[tid] = max( shared[tid], shared[tid+s]);
    	}
      __syncthreads();
   	}

   	if(tid == 0){
      	d_result[ blockIdx.x ] = shared[tid];
   	}
}


__global__ void radix_predicate(   unsigned int * const d_input,
                                   //unsigned int* const d_position,
                                   unsigned int * const d_predicate,
                                   int current_bit,
                                   int size
                                   ){
   
	int index = threadIdx.x + blockDim.x*blockIdx.x;
	if(index >= size ){ return; }

	unsigned int x = d_input[index];
	x >>= current_bit;
	d_predicate[index] = x&1;
}


__global__ void blelloch1_reduction_scan( unsigned int * const d_elements,
							   			  int numElems
										){

   	extern __shared__ unsigned int shared[];
   	int index = threadIdx.x + blockDim.x * blockIdx.x;
   	int tid = threadIdx.x;
   	if( index < size ){
		shared[tid] = d_elements[index];
	}
   	__syncthreads();


  	for( int s = blockDim.x/2 ; s > 0 ; s >>= 1 ){
    	if( tid < s && index < size ){
			shared[tid] = max( shared[tid], shared[tid+s]);
    	}
    	__syncthreads();
   	}


   	//if( index == 0 ){

   	//}


}



__global__ void scan_inplace_threads(
									unsigned int * const d_elements,
									int numElems
									){
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;
	int bos = blockDim.x * blockIdx.x;

	for( int s = 1 ; s < blockDim.x ; s <<= 1 ){
		unsigned int val = 0;
		int spot = tid - s;
		if( spot >= 0 && index < numElems){
	 		val = d_elements[spot+bos];
		}
		__syncthreads();
		if( spot >= 0 && index < numElems){
			d_elements[index] += val;
		}
		__syncthreads();
	}
}

__global__ void scan_get_block_sum( 
								unsigned int * const d_scanned_elements,//expected to be an inclusive scan
								unsigned int * const d_blocksums,
								int numElems
							 ){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	
	if( threadIdx.x + 1 == blockDim.x && index < numElems || index+1 == numElems ){ 
		d_blocksums[blockIdx.x] = d_scanned_elements[index];
	}
}


__global__ void scan_add_block_sum(
										unsigned int * const d_elements,
										unsigned int * const d_blocksums,
										int numElems
									 ){
	int index = threadIdx.x + blockDim.x * blockIdx.x;

	if( index < numElems && index >= blockDim.x ){
		unsigned int local_blocksum = d_blocksums[blockIdx.x - 1];
		d_elements[index] += local_blocksum;
	} 

}

__global__ void scan_polishing1( 
									unsigned int * const d_elems,
									unsigned int * const d_predicate,
									unsigned int * const d_middle,
									int numElems
								){
	int index = threadIdx.x + blockDim.x * blockIdx.x;

	//inclusive to exclusive
	if( index < numElems ){
		unsigned int temp = 0;
		if( index != 0 ){ temp = d_elems[index -1];}
		d_elems[index] = temp;
	}

}


__global__ void scan_polishing2(
									unsigned int * const d_elems,
									unsigned int * const d_predicate,
									unsigned int * const d_middle,
									int numElems
								){
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if(index < numElems){
		unsigned int identity = d_middle[0];
		d_elems[index] += identity;
	}

}

__global__ void scan_polishing3(
									unsigned int * const d_elems,
									unsigned int * const d_predicate,
									unsigned int * const d_middle,
									int numElems									
								){
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if( index == numElems - 1 && index < numElems){ 
		d_middle[0] = d_elems[index];
		d_middle[0] += d_predicate[index];
	}
}


__global__ void radix_invert_predicate( unsigned int * const d_predicate,
                                        int size){
	int index = threadIdx.x + blockDim.x*blockIdx.x;
	if( index >= size ){ return;}
	d_predicate[index] == 0 ? d_predicate[index] = 1 : d_predicate[index] = 0;
}


__global__ void radix_reposition( 
							 unsigned int* const d_position1,
						     unsigned int* const d_position2,
                             unsigned int* const d_predicate,
                             unsigned int* const d_values,
                             unsigned int* const d_value_buffer,
                             unsigned int* const d_inputPos,//check
                             unsigned int* const d_pos_buffer,
                             int size
                             	){

	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if( index < size ){
		unsigned int displacement_index = 0;
		if(d_predicate[index] == 1){
			displacement_index = d_position2[index];
		}
		else{
			displacement_index = d_position1[index];
		}
		d_value_buffer[displacement_index] =   d_values[index];
		d_pos_buffer[displacement_index]   = d_inputPos[index];
	}
	// after this copy value_buffer to values(after synchronization)
}

__global__ void regular_relocate1(
									unsigned int* const d_values,
									unsigned int* const d_position,
									unsigned int* const d_value_buffer,
									int size
								){
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if( index < size ){
		d_value_buffer[d_position[index]] = d_values[index];
	}
}

__global__ void regular_relocate2(
									unsigned int* const d_values,
									unsigned int* const d_position,
									unsigned int* const d_value_buffer,
									int size
								){
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if( index < size ){
		d_values[index] = d_value_buffer[index];
		d_position[index] = index;
	}
}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); } // copied from stack overflow, used to check gpuError codes while debugging.
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
   }
}

#define MAX_BLOCKSZ 1024

using namespace std;


//Functions to calculate gridSize and blockSize consistently.
int getBlockSize(int numElems){
	if(numElems > MAX_BLOCKSZ){
		return MAX_BLOCKSZ;
	}
	else{
		return numElems;
	}
}

int getGridSize(int numElems){
	return (numElems + MAX_BLOCKSZ -1)/MAX_BLOCKSZ;
}



//find max reduce works by reducing blocks then reducing the result of the blocks.
int find_max_reduce( unsigned int* const d_position , int numElems ){
	int blockSize = getBlockSize(numElems);
	int gridSize = getGridSize(numElems);
	int outFElems = gridSize * sizeof(unsigned int); //filesize of the output from one iteration of kernel

	unsigned int * d_result; //pointer to where result of max reduce kernel can be stored
	hipMalloc( (void**)&d_result, outFElems ); 
	unsigned int * d_incopy; //a copy of input
	hipMalloc( (void**)&d_incopy, sizeof(unsigned int)*numElems );
	hipMemcpy( d_incopy, d_position, sizeof(unsigned int)*numElems, hipMemcpyDeviceToDevice );

	int shareFSize = blockSize * sizeof(unsigned int);
	
	int s = numElems; 
	do{
		int current_gridSz = getGridSize(s);
		max_reduce<<< current_gridSz, blockSize, shareFSize>>>( d_incopy, d_result, s );
		hipDeviceSynchronize();
		hipMemcpy( d_incopy, d_result, current_gridSz*sizeof(unsigned int), hipMemcpyDeviceToDevice);

		std::cout << "max_reduce called" << std::endl; //@test

		s /= blockSize;
	}while( s > blockSize );
	unsigned int result = 0;
	hipMemcpy( &result, d_result , sizeof(unsigned int), hipMemcpyDeviceToHost );
	return result;
}

#include <queue>
//Scan arbitrary is a function that utilises a series of kernel calls to run a Hillis and Steele scan
//It scans on block level with sync threads, then takes the block sums from each block and runs
//another scan on that in a loop until there number of elements remaining can fit into a single block.
void scan_arbitrary( 
						unsigned int * const d_elements,
						unsigned int * const d_predicate,
						unsigned int * const d_middle,
						int numElems ){

	cout << "====arbitrary scan====" << endl;
	int gridSize = getGridSize(numElems);
	int blockSize = getBlockSize(numElems);

	int cycles = 0;
	for( int tempGridSize = gridSize, 
			 tempNumElems = numElems 
		    ;tempNumElems > 1
		    ;tempNumElems = tempGridSize, 
		 	 tempGridSize = getGridSize(tempNumElems),
			 cycles++
	);//After the for loop cycles ends up being the correct size
	//running this loop for i < cycles will loop the corresponding number of cycles in this loop.

	vector<unsigned int*> D_scan_targets(cycles+1);
	vector<int> blocksum_numelems(cycles+1);
	scan_inplace_threads<<<gridSize,blockSize>>>( d_elements, numElems );
	hipDeviceSynchronize();
	D_scan_targets[0] = d_elements;
	blocksum_numelems[0] = numElems; 

	for( int i = 0, 
		     tempGridSize = gridSize,
		     tempNumElems = numElems
		    ;i < cycles
			;i++
	){
		unsigned int * d_blockscan;
		gpuErrchk( hipMalloc( (void**)&d_blockscan , gridSize*sizeof(unsigned int) ) );
		
		scan_get_block_sum<<<tempGridSize,blockSize>>>(D_scan_targets[i], d_blockscan, tempNumElems);
			hipDeviceSynchronize();

		//resize dimensions to calculate blocksums
		tempNumElems = tempGridSize ;
		tempGridSize = getGridSize(tempNumElems) ;
		if( tempNumElems > 1 ){

			scan_inplace_threads<<<tempGridSize,blockSize>>>( d_blockscan, tempNumElems );
				hipDeviceSynchronize();

			D_scan_targets[i+1] = d_blockscan;
			blocksum_numelems[i+1] = tempNumElems;

		}

	}

	for(int i =  cycles - 1 ; i > 0 ; i-- ){
		int tempGridSize = getGridSize( blocksum_numelems[i-1] );
		
		scan_add_block_sum<<<tempGridSize,blockSize>>>( D_scan_targets[i-1], D_scan_targets[i], blocksum_numelems[i-1] );
		hipDeviceSynchronize();
		//Consider freeing memory here
	}


	for( int i = 1 ; i < cycles ; i++ ){
		hipFree(D_scan_targets[i]);// free all the elements except the first since it is used again after function end.
	}

	scan_polishing1<<<gridSize,blockSize>>>( d_elements, d_predicate, d_middle, numElems);
			hipDeviceSynchronize();
	scan_polishing2<<<gridSize,blockSize>>>( d_elements, d_predicate, d_middle, numElems);
			hipDeviceSynchronize();
	scan_polishing3<<<gridSize,blockSize>>>( d_elements, d_predicate, d_middle, numElems);
			hipDeviceSynchronize();
	
}


//Note the use of global memory in the reposition phases.
//I wasn't sure how else to ensure that relocations happened in sync so I ran them in separate kernels so that
// each step was guaranteed to be completed before moving onto the next ie-  block level synchronisation was needed.
void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_numbers,
               unsigned int* const d_position1,
               const size_t numElems)
{

	const int filesize = numElems*sizeof(unsigned int);

	unsigned int* d_predicate;
	unsigned int* d_position2;
	unsigned int* d_numbers_buffer;
	unsigned int* d_inputPos_buffer;

	int blockSize =  getBlockSize(numElems);
	int gridSize = getGridSize(numElems);


	gpuErrchk(hipMalloc((void**)&d_predicate, filesize ));
	gpuErrchk(hipMalloc((void**)&d_position2, filesize ));
	gpuErrchk(hipMalloc((void**)&d_numbers_buffer, filesize ));
	gpuErrchk(hipMalloc((void**)&d_inputPos_buffer, filesize ));

	//move all elements such that the position vector is in sorted order.
	regular_relocate1<<<gridSize,blockSize>>>(d_inputVals, d_inputPos, d_numbers_buffer, numElems);
		hipDeviceSynchronize();
	regular_relocate2<<<gridSize,blockSize>>>(d_inputVals, d_inputPos, d_numbers_buffer, numElems);
		hipDeviceSynchronize();

	//set scan destination to 0;
	gpuErrchk(hipMemset( d_predicate, 0, filesize));
	//gpuErrchk(hipMemset( d_numbers_buffer, 0 , filesize ));
	gpuErrchk(hipMemcpy ( d_position1, d_inputPos, filesize, hipMemcpyDeviceToDevice ));
	gpuErrchk(hipMemcpy ( d_position2, d_inputPos, filesize, hipMemcpyDeviceToDevice ));
	gpuErrchk(hipMemcpy ( d_numbers, d_inputVals, filesize, hipMemcpyDeviceToDevice ));//Values changed after calculating most significant element.


	unsigned int * d_middle;
	gpuErrchk(hipMalloc((void**)&d_middle, sizeof(unsigned int) ));
	unsigned int bitsig = find_max_reduce( d_numbers, numElems);
	cout<< "bitsig:" << bitsig << endl;
	if( bitsig <= UINT_MAX/2 ){ bitsig <<= 1; }
	else{ bitsig = UINT_MAX; }


	for ( int current_bit = 0 ; (bitsig >> current_bit) > 1 ; current_bit++  ){

		cout << "currentbit:" << current_bit << endl;
		gpuErrchk(hipMemset(d_middle,0, sizeof(unsigned int)));
			hipDeviceSynchronize();

		radix_predicate<<<gridSize,blockSize>>>(d_numbers, d_predicate, current_bit, numElems);
			hipDeviceSynchronize();

		radix_invert_predicate<<<gridSize,blockSize>>>(d_predicate, numElems);
			hipDeviceSynchronize();

		hipMemcpy(d_position1, d_predicate, filesize, hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();

		scan_arbitrary(d_position1 ,d_predicate, d_middle, numElems);
			hipDeviceSynchronize();

		radix_invert_predicate<<<gridSize,blockSize>>>(d_predicate, numElems);
			hipDeviceSynchronize();

		hipMemcpy(d_position2, d_predicate, filesize, hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();

		scan_arbitrary(d_position2 , d_predicate, d_middle, numElems);
			hipDeviceSynchronize();

		radix_reposition<<<gridSize,blockSize>>>(d_position1, d_position2, d_predicate, d_numbers, d_numbers_buffer, d_inputPos, d_inputPos_buffer, numElems );
			hipDeviceSynchronize();
		hipMemcpy(d_numbers, d_numbers_buffer, filesize, hipMemcpyDeviceToDevice );
		hipMemcpy(d_inputPos, d_inputPos_buffer, filesize, hipMemcpyDeviceToDevice );
			hipDeviceSynchronize();
	}

	gpuErrchk( hipMemcpy( d_inputVals, d_numbers, filesize, hipMemcpyDeviceToDevice ));
	gpuErrchk( hipMemcpy( d_position1, d_inputPos, filesize, hipMemcpyDeviceToDevice ));

	hipFree(d_predicate);
	hipFree(d_position2);
	hipFree(d_middle);
}


int main(int argc, char * argv[]){
	if(argc !=  2){
		cout <<   "usage: " << argv[0] << "<filename>" << endl;
	}
	else{
		string line;
		ifstream myfile(argv[1]);
		int size = 0;
		int filesize = 0;
		int lines = 0;
		unsigned int * h_numbers;
		unsigned int * h_position;
		if( myfile.is_open()){
			getline(myfile,line);
			size = atoi(line.c_str());//first line of file is assumed to show number of elements in file.
			filesize = sizeof(int)*size;
			h_numbers = (unsigned int *)malloc(filesize);
			h_position = (unsigned int *)malloc(filesize);
			int i = 0;
			while(getline(myfile, line)){
				 h_numbers[i] = atoi(line.c_str());
				 h_position[i] = i;
				 lines++;
				 i++;
			}
			myfile.close();
		}
		else {
			cout << "Sorry mate, can't load file" << endl;
			return 0;
		}
			//allocate memory for device
		unsigned int * d_numbers;
		unsigned int * d_position;
		unsigned int * d_out_numbers;
		unsigned int * d_out_position;

		gpuErrchk( hipMalloc((void**)&d_numbers, filesize));
		hipDeviceSynchronize();
		gpuErrchk( hipMalloc((void**)&d_position, filesize));
		hipDeviceSynchronize();
		gpuErrchk( hipMalloc((void**)&d_out_numbers, filesize));
		hipDeviceSynchronize();
		gpuErrchk( hipMalloc((void**)&d_out_position, filesize));
		hipDeviceSynchronize();

		//copy memory host to device
		gpuErrchk( hipMemcpy( d_numbers, h_numbers, filesize , hipMemcpyHostToDevice ));
		hipDeviceSynchronize();
		gpuErrchk( hipMemcpy( d_position, h_position, filesize, hipMemcpyHostToDevice ));
		hipDeviceSynchronize();

		//run sort
		your_sort(d_numbers, d_position, d_out_numbers, d_out_position, lines );
		//hipDeviceSynchronize();

		gpuErrchk( hipMemcpy( h_numbers, d_out_numbers ,filesize, hipMemcpyDeviceToHost));
		gpuErrchk( hipMemcpy (h_position, d_position, filesize, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		ofstream sortedfile("sorted");
	    if( sortedfile.is_open()){
	      for(int i = 0 ; i < lines ; i++){
	        sortedfile << std::to_string(h_numbers[i]) << "         " <<   to_string(h_position[i])   <<"\n";
	      }
	    }
		std::cout << "finished" <<std::endl;

		return 0;

	}

}
