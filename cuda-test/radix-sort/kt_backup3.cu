
#include "assert.h"

#include <hip/hip_runtime.h> //used for assert
#include <iostream>

#include <vector>

#include <fstream>
#include <string>
#include <climits>

//backup version stored as kt_backup.cu

//#define MAXSHID2 2*blockDim.x-1
#define INNERGRID gridDim.x/blockDim.x
#define MAX_BLOCKSZ 512

__device__ int d_getBlockSize(int numElems){
  if(numElems > MAX_BLOCKSZ){
    return MAX_BLOCKSZ;
  }
  else{
    return numElems;
  }
}

__device__ int d_getGridSize(int numElems){
  return (numElems + MAX_BLOCKSZ -1)/MAX_BLOCKSZ;
}

__global__ void bookmark_blocksum( 
  unsigned int* d_elements, 
           int* d_bookmarks,
            int blockSumNumElems,
            int depth)
{
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if( depth > 0 && 2*index+1 < blockSumNumElems){
    int blocks_mark = d_bookmarks[depth-1];
    int sum_mark = d_bookmarks[depth];
    //d_elements[blocks_mark + index] = 0;
    d_elements[blocks_mark + 2*index] +=  d_elements[sum_mark + blockIdx.x];
    d_elements[blocks_mark + 2*index + 1 ] +=  d_elements[sum_mark + blockIdx.x];
  }
  else{
    //throw some kind of error. consider using assert or something here instead.
  }

}


__global__ void blelloch_threadsum(
  unsigned int* d_elements,
            int numElems,
            int nextmark)
{
    extern __shared__ unsigned int shared[];
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    if( 2*index+1 < numElems  ){
      shared[2*tid] = d_elements[2*index];
      shared[2*tid+1] = d_elements[2*index + 1 ];
    }
    __syncthreads();

    int s = 1;
    for( int d = blockDim.x ; d > 0  ; d >>= 1 ){
      __syncthreads();
      if( tid < d){ 
        unsigned int addn = shared[s*(2*tid + 1) - 1];
        unsigned int dest = shared[s*(2*tid + 2) - 1];
        shared[s*(2*tid + 2) - 1] = dest + addn;
      }
      s *= 2;
    }

    if(gridDim.x > 1){
      d_elements[nextmark + blockIdx.x] = shared[2*blockDim.x - 1];
    }
    
    if(tid == 0  ){
      shared[2*blockDim.x - 1] = 0;
    }

    for( int d = 1 ; d < 2*blockDim.x ; d *= 2 ){
      s >>= 1;
      __syncthreads();
      if(tid < d){
        unsigned int addn = shared[s*(2*tid + 1) - 1];
        unsigned int dest = shared[s*(2*tid + 2) - 1];
        shared[s*(2*tid + 1) - 1] = dest;
        shared[s*(2*tid + 2) - 1] = dest + addn;
      }
    }
    __syncthreads();
    
    d_elements[2*index] = shared[2*tid];
    d_elements[2*index + 1] = shared[2*tid + 1];
    __syncthreads();
    
}


__global__ void hillisteel_tailsum( //only use this for the last block of an array.
                  unsigned int * const d_elements,
                                   int numElems,
                                   int nextmark)
{
  int tid = threadIdx.x;
  extern __shared__ unsigned int shared[];

  if(tid < numElems){
    shared[tid] = d_elements[tid];
  }
  __syncthreads();

  for( int s = 1 ; s < blockDim.x ; s <<= 1 ){
    unsigned int val = 0;
    int spot = tid - s;
    if( spot >= 0 && tid < numElems ){
      val = shared[spot];
    }
    __syncthreads();
    if( spot >= 0 && tid < numElems ){
      shared[tid] += val;
    }
    __syncthreads();
  }
  d_elements[tid] = shared[tid];

}


using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); } // copied from stack overflow, used to check gpuError codes while debugging.
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
   }
}


int getBlockSize(int numElems){
  if(numElems > MAX_BLOCKSZ){
    return MAX_BLOCKSZ;
  }
  else{
    return numElems;
  }
}

int getGridSize(int numElems){
  return (numElems + MAX_BLOCKSZ -1)/MAX_BLOCKSZ;
}

void arbitrary_scan( unsigned int * h_elements, int numElems){
    //allocate memory for device
    unsigned int * d_elements;
    int shareSize = 2*MAX_BLOCKSZ * sizeof(unsigned int);

    //this section is used to set up starting conditions for the kernel
    //as well as information for it to continue - eg the bookmarks are used
    //by the kernel to find indexes for the blocksumming stages of the scan.
    int bookmarks_sz =  128; //@note this is probably un-necessarily large, with a block size of 512, this will only be resizes if numElems ~= 512^128, which is huge.
    vector<int> h_bookmarks(bookmarks_sz, 0); //@improve? 128 is arbitrarily selected, since the final size is unknown, and a resize every single loop is not preferable. Hence a resize every '32' is used.
    int workneeded = numElems;
    int depth = 0;
    int tmp_elems = (numElems+(MAX_BLOCKSZ*2) -1)/(MAX_BLOCKSZ*2);
    while(tmp_elems > 1 ){
      depth++;
      if( depth+1 >= bookmarks_sz-1 ){ //if too deep, increase size to allow for more bookmark entries. 
        bookmarks_sz += 128;
        h_bookmarks.resize(bookmarks_sz); 
      }
      h_bookmarks[depth] = workneeded; //note depth 0 case was not specifically handled by loop, but is set to 0 by initialisation of the vector, and is simply skipped by loop.
      workneeded += tmp_elems;
      tmp_elems = (tmp_elems+(MAX_BLOCKSZ*2) - 1)/(MAX_BLOCKSZ*2);
    }
    h_bookmarks[depth+1] = workneeded;

    int worksize = (workneeded+1) * sizeof(unsigned int); //plus 1 since sometimes the kernel will operate on an index 1 more than max without checking.
    int filesize = numElems * sizeof(unsigned int);

    gpuErrchk( hipMalloc((void**)&d_elements, worksize));
    gpuErrchk( hipMemset( d_elements, 0, worksize));
    gpuErrchk( hipMemcpy( d_elements, h_elements, filesize , hipMemcpyHostToDevice )); //change back to filesize@test
    
    
    for(  int i = 0,
              it_numElems = numElems
          ; i <= depth ; i++ )
    {
      int it_remSz  = it_numElems%(2*MAX_BLOCKSZ);
      int it_gridSz = it_numElems/(2*MAX_BLOCKSZ);
      int remMark   = h_bookmarks[i+1] - it_remSz;
      int nextmark  = h_bookmarks[i+1] - h_bookmarks[i];
      std::cout << "h_bookmarks[" << i+1 << "]" <<h_bookmarks[i+1] << std::endl;
      std::cout << "remMark:"   << remMark   << std::endl;
      std::cout << "it_remSz:"  << it_remSz  << std::endl;
      std::cout << "it_gridSz:" << it_gridSz << std::endl;
      std::cout << "shareSize:" << shareSize << std::endl;
      if( it_gridSz ){ blelloch_threadsum<<<it_gridSz, MAX_BLOCKSZ, shareSize>>>( d_elements + h_bookmarks[i] , it_numElems, nextmark ); }
      //cudaDeviceSynchronize();
      if( it_remSz  ){ hillisteel_tailsum<<<1, it_remSz,shareSize>>>( d_elements + remMark, it_remSz, nextmark ); }
      hipDeviceSynchronize();
      
      it_numElems = it_gridSz + (it_remSz >= 1) ;
      it_gridSz   = it_numElems;
    }
    //blelloch_threadsum<<<gridSize,blockSize,shareSize>>>(d_elements, d_bookmarks, numElems, 0)  ;
    /*
    for( int i = depth ; i >= 1 ; i--){
      int it_numElems  = h_bookmarks[i] - h_bookmarks[i-1],
          it_blockSz   = getBlockSize(it_numElems),
          it_gridSz    = getGridSize (it_numElems);
      blelloch_blocksum<<<it_gridSz,it_blockSz>>>(d_elements, d_bookmarks,it_numElems, i);
      cudaDeviceSynchronize();
    }*/
    unsigned int* h_test_elements = (unsigned int*)malloc(worksize*sizeof(unsigned int));
    gpuErrchk( hipMemcpy( h_test_elements, d_elements, worksize, hipMemcpyDeviceToHost)); //@test - used for checking memory in cuda-gdb.
    gpuErrchk( hipMemcpy( h_elements, d_elements, filesize, hipMemcpyDeviceToHost)); 
}


int main(int argc, char * argv[]){
  if(argc !=  2){
    cout <<   "usage: " << argv[0] << "<filename>" << endl;
  }
  else{
    string line;
    ifstream myfile(argv[1]);
    int size = 0;
    int filesize = 0;
    int lines = 0;
    unsigned int * h_numbers;

    if( myfile.is_open()){
      getline(myfile,line);
      size = atoi(line.c_str());//first line of file is assumed to show number of elements in file.
      filesize = sizeof(int)*size;
      h_numbers = (unsigned int *)malloc(filesize);
      int i = 0;
      while(getline(myfile, line)){
         h_numbers[i] = atoi(line.c_str());

         lines++;
         i++;
      }
      myfile.close();
    }
    else {
      cout << "Sorry mate, can't load file" << endl;
      return 0;
    }

    arbitrary_scan(h_numbers, lines);

    ofstream sortedfile("sorted");
      if( sortedfile.is_open()){
        for(int i = 0 ; i < lines; i++){
          sortedfile << std::to_string(h_numbers[i]) <<"\n";
        }
      }
    std::cout << "finished" <<std::endl;

    return 0;

  }
}