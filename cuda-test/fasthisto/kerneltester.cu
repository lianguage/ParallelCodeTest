#include "hip/hip_runtime.h"

#include "assert.h"

#include <hip/hip_runtime.h> //used for assert
#include <iostream>

#include <vector>

#include <fstream>
#include <string>
#include <climits>

//backup version stored as kt_backup.cu
#define MAX_BLOCKSZ 512

__global__ void bookmark_blocksum( 
  unsigned int* d_elements, 
            int blockSumNumElems,
            int nextmark)
{
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if( 2*index+1 < blockSumNumElems){
    //d_elements[blocks_mark + index] = 0;
    d_elements[2*index] +=  d_elements[nextmark + blockIdx.x];
    d_elements[2*index + 1] +=  d_elements[nextmark + blockIdx.x];
  }
  else{
    //@todo throw some kind of error. consider using assert or something here instead.
  }
}


__global__ void blelloch_threadsum( 
  unsigned int* d_elements,
            int numElems,
            int nextmark)
{
    //@note @unexplained behavior, see blelloch-weirdbug.cu
    extern __shared__ unsigned int shared[];
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    if( 2*index+1 < numElems  ){
      shared[2*tid] = d_elements[2*index];
      shared[2*tid+1] = d_elements[2*index + 1 ];
    }
    __syncthreads();

    int s = 1;
    for( int d = blockDim.x ; d > 0  ; d >>= 1 ){
      __syncthreads();
      if( tid < d){ 
        unsigned int addn = shared[s*(2*tid + 1) - 1];
        unsigned int dest = shared[s*(2*tid + 2) - 1];
        shared[s*(2*tid + 2) - 1] = dest + addn;
      }
      s *= 2;
    }

    if(tid == 0  ){
      d_elements[nextmark + blockIdx.x] = shared[2*blockDim.x - 1];
      shared[2*blockDim.x - 1] = 0;
    }

    for( int d = 1 ; d < 2*blockDim.x ; d *= 2 ){
      s >>= 1;
      __syncthreads();
      if(tid < d){
        unsigned int addn = shared[s*(2*tid + 1) - 1];
        unsigned int dest = shared[s*(2*tid + 2) - 1];
        shared[s*(2*tid + 1) - 1] = dest;
        shared[s*(2*tid + 2) - 1] = dest + addn;
      }
    }
    __syncthreads();
    
    if( 2*index+1 < numElems  ){
      d_elements[2*index] = shared[2*tid];
      d_elements[2*index + 1] = shared[2*tid + 1];
    }
}


__global__ void hillisteel_tailsum( //only use this for the last block of an array.
                  unsigned int * const d_elements,
                                   int numElems,
                                   int nextmark)
{
  int tid = threadIdx.x;
  extern __shared__ unsigned int shared[];

  if(tid < numElems){
    shared[tid] = d_elements[tid];
  }
  __syncthreads();

  for( int s = 1 ; s < blockDim.x ; s <<= 1 ){
    unsigned int val = 0;
    int spot = tid - s;
    if( spot >= 0 && tid < numElems ){
      val = shared[spot];
    }
    __syncthreads();
    if( spot >= 0 && tid < numElems ){
      shared[tid] += val;
    }
    __syncthreads();
  }
  if( tid+1 < numElems ){
    d_elements[tid+1] = shared[tid];
  }
  else if( tid == numElems-1){
    //d_elements[nextmark] = shared[tid];
    d_elements[0] = 0;
  }

}


using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); } // copied from stack overflow, used to check gpuError codes while debugging.
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
   }
}


int getBlockSize(int numElems){
  if(numElems > MAX_BLOCKSZ){
    return MAX_BLOCKSZ;
  }
  else{
    return numElems;
  }
}

int getGridSize(int numElems){
  return (numElems + MAX_BLOCKSZ -1)/MAX_BLOCKSZ;
}

void arbitrary_scan( unsigned int * h_elements, int numElems){
    //allocate memory for device
    unsigned int * d_elements;
    int shareSize = 2*MAX_BLOCKSZ * sizeof(unsigned int);

    //this section is used to set up starting conditions for the kernel
    //as well as information for it to continue - eg the bookmarks are used
    //by the kernel to find indexes for the blocksumming stages of the scan.
    int bookmarks_sz =  128; //@note this is probably un-necessarily large, with a block size of 512, this will only be resizes if numElems ~= 512^128, which is huge.
    vector<int> h_bookmarks(bookmarks_sz, 0); //@improve? 128 is arbitrarily selected, since the final size is unknown, and a resize every single loop is not preferable. Hence a resize every '32' is used.
    int workneeded = numElems;
    int depth = 0;
    int tmp_elems = (numElems+(MAX_BLOCKSZ*2) -1)/(MAX_BLOCKSZ*2);
    while(tmp_elems > 1 ){
      depth++;
      if( depth+1 >= bookmarks_sz-1 ){ //if too deep, increase size to allow for more bookmark entries. 
        bookmarks_sz += 128;
        h_bookmarks.resize(bookmarks_sz); 
      }
      h_bookmarks[depth] = workneeded; //@note depth 0 case was not specifically handled by loop, but is set to 0 by initialisation of the vector, and is simply skipped by loop.
      workneeded += tmp_elems;
      tmp_elems = (tmp_elems+(MAX_BLOCKSZ*2) - 1)/(MAX_BLOCKSZ*2);
    }
    h_bookmarks[depth+1] = workneeded;

    int worksize = (workneeded+1) * sizeof(unsigned int); //plus 1 since sometimes the kernel will operate on an index 1 more than max without checking.
    int filesize = numElems * sizeof(unsigned int);

    unsigned int * d_elements;
    gpuErrchk( hipMalloc((void**)&d_elements, worksize));
    //gpuErrchk( hipMemset( d_elements, 0, worksize));
    gpuErrchk( hipMemcpy( d_elements, h_elements, filesize , hipMemcpyHostToDevice )); //change back to filesize@test
    
    unsigned int* h_test_elements = (unsigned int*)malloc(worksize*sizeof(unsigned int));//@test
    for(  int i = 0,
              it_numElems = numElems
          ; i <= depth ; i++ )
    {
      
      //Set up initial conditions
      int it_remSz  = it_numElems%(2*MAX_BLOCKSZ),
          it_gridSz = it_numElems/(2*MAX_BLOCKSZ),
          remMark   = h_bookmarks[i+1] - it_remSz,
          nextmark  = h_bookmarks[i+1] - h_bookmarks[i];
      //run the relevant kernel if conditions are correct
      if( it_gridSz ){ blelloch_threadsum<<<it_gridSz, MAX_BLOCKSZ, shareSize>>>( d_elements + h_bookmarks[i] , it_numElems, nextmark ); }
      gpuErrchk( hipMemcpy( h_test_elements, d_elements, worksize, hipMemcpyDeviceToHost)); //@test - used for checking memory in cuda-gdb.
      if( it_remSz  ){ hillisteel_tailsum<<<1, it_remSz,shareSize>>>( d_elements + remMark, it_remSz, nextmark + it_gridSz ); }
      gpuErrchk( hipMemcpy( h_test_elements, d_elements, worksize, hipMemcpyDeviceToHost)); //@test - used for checking memory in cuda-gdb.
      hipDeviceSynchronize();

      //increment the elements in loop
      it_numElems = it_gridSz + (it_remSz >= 1) ;
      it_gridSz   = it_numElems;
    }
    
    for( int i = depth ; i > 0 ; i--){
      int it_numElems  = h_bookmarks[i] - h_bookmarks[i-1],
          it_blockSz   = getBlockSize(it_numElems),
          it_gridSz    = getGridSize (it_numElems),
          nextmark     = h_bookmarks[i] - h_bookmarks[i-1];
      bookmark_blocksum<<<it_gridSz,it_blockSz>>>(d_elements + h_bookmarks[i-1], it_numElems, nextmark);
      hipDeviceSynchronize();
    }
    gpuErrchk( hipMemcpy( h_test_elements, d_elements, worksize, hipMemcpyDeviceToHost)); //@test - used for checking memory in cuda-gdb.
    gpuErrchk( hipMemcpy( h_elements, d_elements, filesize, hipMemcpyDeviceToHost)); 
}


int main(int argc, char * argv[]){
  if(argc !=  2){
    cout <<   "usage: " << argv[0] << "<filename>" << endl;
  }
  else{
    string line;
    ifstream myfile(argv[1]);
    int size = 0;
    int filesize = 0;
    int lines = 0;
    unsigned int * h_numbers;

    if( myfile.is_open()){
      getline(myfile,line);
      size = atoi(line.c_str());//first line of file is assumed to show number of elements in file.
      filesize = sizeof(int)*size;
      h_numbers = (unsigned int *)malloc(filesize);
      int i = 0;
      while(getline(myfile, line)){
         h_numbers[i] = atoi(line.c_str());

         lines++;
         i++;
      }
      myfile.close();
    }
    else {
      cout << "Sorry mate, can't load file" << endl;
      return 0;
    }

    arbitrary_scan(h_numbers, lines);

    ofstream sortedfile("sorted");
      if( sortedfile.is_open()){
        for(int i = 0 ; i < lines; i++){
          sortedfile << std::to_string(h_numbers[i]) <<"\n";
        }
      }
    std::cout << "finished" <<std::endl;

    return 0;

  }
}