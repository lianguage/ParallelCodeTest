
#include "assert.h"

#include <hip/hip_runtime.h> //used for assert
#include <iostream>

#include <vector>
#include <deque>

#include <fstream>
#include <string>
#include <climits>

//backup version stored as kt_backup.cu
#define MAX_BLOCKSZ 512

__global__ void bookmark_blocksum(  

  unsigned int* d_elements,  
            int blockSumNumElems, 
            int nextmark
)
{
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if( 2*index+1 < blockSumNumElems){
    //d_elements[blocks_mark + index] = 0;
    d_elements[2*index] +=  d_elements[nextmark + blockIdx.x];
    d_elements[2*index + 1] +=  d_elements[nextmark + blockIdx.x];
  }
  else{
    
    //@todo throw some kind of error. consider using assert or something here instead.

  }
}


__global__ void blelloch_threadsum(

  unsigned int* d_elements, 
            int numElems, 
            int nextmark){
    //@note @unexplained behavior, see blelloch-weirdbug.cu
    extern __shared__ unsigned int shared[];
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    if( 2*index+1 < numElems  ){
      shared[2*tid] = d_elements[2*index];
      shared[2*tid+1] = d_elements[2*index + 1 ];
    }
    __syncthreads();

    int s = 1;
    for( int d = blockDim.x ; d > 0  ; d >>= 1 ){
      __syncthreads();
      if( tid < d){ 
        unsigned int addn = shared[s*(2*tid + 1) - 1];
        unsigned int dest = shared[s*(2*tid + 2) - 1];
        shared[s*(2*tid + 2) - 1] = dest + addn;
      }
      s *= 2;
    }

    if(tid == 0  ){
      d_elements[nextmark + blockIdx.x] = shared[2*blockDim.x - 1];
      shared[2*blockDim.x - 1] = 0;
    }

    for( int d = 1 ; d < 2*blockDim.x ; d *= 2 ){
      s >>= 1;
      __syncthreads();
      if(tid < d){
        unsigned int addn = shared[s*(2*tid + 1) - 1];
        unsigned int dest = shared[s*(2*tid + 2) - 1];
        shared[s*(2*tid + 1) - 1] = dest;
        shared[s*(2*tid + 2) - 1] = dest + addn;
      }
    }
    __syncthreads();
    
    if( 2*index+1 < numElems  ){
      d_elements[2*index] = shared[2*tid];
      d_elements[2*index + 1] = shared[2*tid + 1];
    }
}

//only use this for the last block of an array.
__global__ void hillisteel_tailsum(

  unsigned int * const d_elements, 
                   int numElems, 
                   int nextmark
)
{

  int tid = threadIdx.x;
  extern __shared__ unsigned int shared[];

  if(tid < numElems){
    shared[tid] = d_elements[tid];
  }
  __syncthreads();

  for( int s = 1 ; s < blockDim.x ; s <<= 1 ){
    unsigned int val = 0;
    int spot = tid - s;
    if( spot >= 0 && tid < numElems ){
      val = shared[spot];
    }
    __syncthreads();
    if( spot >= 0 && tid < numElems ){
      shared[tid] += val;
    }
    __syncthreads();
  }
  if( tid+1 < numElems ){
    d_elements[tid+1] = shared[tid];
  }
  else if( tid == numElems-1){
    //d_elements[nextmark] = shared[tid];
    d_elements[0] = 0;
  }

}

__global__ void radix_predicate( 

  const unsigned int * const d_input,
        unsigned int * const d_odds,
        unsigned int * const d_evens,
                         int current_bit,
                         int numElems
)
{
  int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index >= numElems ){ return; } //@note early return is ok, because __syncthreads() was not used.

  unsigned int x = d_input[index];
  x >>= current_bit;
  d_odds [index] =   x&1 ;
  d_evens[index] = !(x&1);
}

__global__ void add_1d( unsigned int * d_array,
                          unsigned int value,
                                   int numElems)
{
  
}

/*
__glboal__ void invert_predicate( unsigned int * const d_predicate, int numElems ){


}*/

__global__ void compact_relocate(

  unsigned int * const d_values,
  unsigned int * const d_newlocation,
                   int totalNumElems
)
{


}



__global__
void yourHisto(const unsigned int* const vals, //INPUT
               unsigned int* const histo, //OUPUT
               int numVals)
{
  //TODO fill in this kernel to calculate the histogram
  //as quickly as possible

  //Although we provide only one kernel skeleton,
  //feel free to use more if it will help you
  //write faster code
  
  
  
}


using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); } // copied from stack overflow, used to check gpuError codes while debugging.
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{

   if (code != hipSuccess) 
   {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
   }

}


int getBlockSize(int numElems){
  if(numElems > MAX_BLOCKSZ){
    return MAX_BLOCKSZ;
  }
  else{
    return numElems;
  }
}

int getGridSize(int numElems){
  return (numElems + MAX_BLOCKSZ -1)/MAX_BLOCKSZ;
}

unsigned int arbitrary_scan( unsigned int * d_input, int numElems)
{
    unsigned int last = 0;
    gpuErrchk( hipMemcpy( &last, d_input + (numElems - 1) , sizeof(unsigned int), hipMemcpyDeviceToHost ));



    //this section is used to set up starting conditions for the kernel
    //as well as information for it to continue - eg the bookmarks are used
    //by the kernel to find indexes for the blocksumming stages of the scan.
    int bookmarks_sz =  128; //@note this is probably un-necessarily large, with a block size of 512, this will only be resizes if numElems ~= 512^128, which is huge.
    vector<int> h_bookmarks(bookmarks_sz, 0); //@improve? 128 is arbitrarily selected, since the final size is unknown, and a resize every single loop is not preferable. Hence a resize every '32' is used.
    int workneeded = numElems;
    int depth = 0;
    int tmp_elems = (numElems+(MAX_BLOCKSZ*2) -1)/(MAX_BLOCKSZ*2);
    while(tmp_elems > 1 ){
      depth++;
      if( depth+1 >= bookmarks_sz-1 ){ //if too deep, increase size to allow for more bookmark entries. 
        bookmarks_sz += 128;
        h_bookmarks.resize(bookmarks_sz); 
      }
      h_bookmarks[depth] = workneeded; //@note depth 0 case was not specifically handled by loop, but is set to 0 by initialisation of the vector, and is simply skipped by loop.
      workneeded += tmp_elems;
      tmp_elems = (tmp_elems+(MAX_BLOCKSZ*2) - 1)/(MAX_BLOCKSZ*2);
    }
    h_bookmarks[depth+1] = workneeded;

    int worksize = (workneeded+1) * sizeof(unsigned int); //plus 1 since sometimes the kernel will operate on an index 1 more than max without checking.
    int filesize = numElems * sizeof(unsigned int);
    int shareSize = 2*MAX_BLOCKSZ * sizeof(unsigned int);
    
    unsigned int * d_elements;
    gpuErrchk( hipMalloc((void**)&d_elements, worksize));
    //gpuErrchk( cudaMemset( d_elements, 0, worksize));
    gpuErrchk( hipMemcpy( d_elements, d_input, filesize , hipMemcpyDeviceToDevice )); //change back to filesize@test
    cout << "1" << endl;
    for(  int i = 0,
              it_numElems = numElems
          ; i <= depth ; i++ )
    {
      
      //Set up initial conditions
      int it_remSz  = it_numElems%(2*MAX_BLOCKSZ),
          it_gridSz = it_numElems/(2*MAX_BLOCKSZ),
          remMark   = h_bookmarks[i+1] - it_remSz,
          nextmark  = h_bookmarks[i+1] - h_bookmarks[i];
      //run the relevant kernel if conditions are correct
      if( it_gridSz ){ blelloch_threadsum<<<it_gridSz, MAX_BLOCKSZ, shareSize>>>( d_elements + h_bookmarks[i] , it_numElems, nextmark ); }
      if( it_remSz  ){ hillisteel_tailsum<<<1, it_remSz,shareSize>>>( d_elements + remMark, it_remSz, nextmark + it_gridSz ); }
      hipDeviceSynchronize();

      //increment the elements in loop
      it_numElems = it_gridSz + (it_remSz >= 1) ;
      it_gridSz   = it_numElems;
    }
    
    for( int i = depth ; i > 0 ; i--){
      int it_numElems  = h_bookmarks[i] - h_bookmarks[i-1],
          it_blockSz   = getBlockSize(it_numElems),
          it_gridSz    = getGridSize (it_numElems),
          nextmark     = h_bookmarks[i] - h_bookmarks[i-1];
      bookmark_blocksum<<<it_gridSz,it_blockSz>>>(d_elements + h_bookmarks[i-1], it_numElems, nextmark);
      hipDeviceSynchronize();
    }

    //Calculate the value that would be taken by summing all elements - ie same as last element in an inclusive scan.
    unsigned int reduced = 0;
    gpuErrchk( hipMemcpy( &reduced, d_elements + (numElems - 1) , sizeof(unsigned int), hipMemcpyDeviceToHost ));
    reduced += last;
    cout << "2" <<endl;
    //copy results to d_input
    gpuErrchk( hipMemcpy( d_input, d_elements, filesize, hipMemcpyDeviceToHost)); 
    return reduced;

}


void computeHistogram(const unsigned int* const d_vals, //INPUT
                      unsigned int* const d_histo, //OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
  //TODO Launch the yourHisto kernel

  //if you want to use/launch more than one kernel,
  //feel free
  //cudaDeviceSynchronize(); //checkCudaErrors(cudaGetLastError());

  //delete[] h_vals;
  //delete[] h_histo;
  //delete[] your_histo;
  //unsigned int * h_histo = (unsigned int*) malloc( numBins * sizeof(unsigned int));//histogram is probably better held on a latency optimised system.



  int filesize  = numElems * sizeof(unsigned int);
  int gridSize  = getGridSize ( numElems );
  int blockSize = getBlockSize( numElems );

  unsigned int * d_odds ;
  unsigned int * d_evens;
  gpuErrchk( hipMalloc( (void**)&d_odds , filesize) );
  gpuErrchk( hipMalloc( (void**)&d_evens, filesize) );
  
  radix_predicate<<<gridSize,blockSize>>>( d_vals, d_evens, d_odds, 1, numElems);

  int numOdds  = arbitrary_scan( d_odds,  numElems );
  int numEvens = arbitrary_scan( d_evens, numElems );



  cout << "odds  :" << numOdds  << endl;
  cout << "evens :" << numEvens << endl;

  hipFree( d_odds );
  hipFree( d_evens);

  std::vector<unsigned int> h_bins   ( numBins, 0 );
  std::deque <unsigned int> h_radices;
  for( int current_bit = 0 ; numBins >> current_bit > 0 ; current_bit++  ){
    //int moo = numBins >> current_bit;//@test
    //cout << "binNum: " << moo << endl;//@test
    
    

  }
  

}

using namespace std;

//test section
int main(int argc, char * argv[]){
  if(argc != 2){
    cout << "usage: " << argv[0] << "<filename>" << endl;
  }
  else{
    string line;
    ifstream myfile(argv[1]);
    int numElems = 0;
    int filesize = 0;
    int lines = 0;
    unsigned int * h_values;
    if( myfile.is_open()){
      getline(myfile,line);
      numElems = atoi(line.c_str());//first line of file is assumed to show number of elements in file.
      filesize = sizeof(unsigned int)*numElems;
      h_values = (unsigned int *)malloc(filesize);
      int i = 0;
      while(getline(myfile, line)){
         h_values[i] = atoi(line.c_str());
         lines++;
         i++;
      }
      myfile.close();
    }
    else {
      cout << "Sorry mate, can't load file" << endl;
      return 0;
    }

    unsigned int * d_values;
    unsigned int * d_histo;
    
    const int NUMBINS = 1024; //defined in main to prevent usage outside of main.
    unsigned int binsize = NUMBINS * sizeof(unsigned int);
    unsigned int * h_histo = (unsigned int *)malloc(binsize);

    gpuErrchk( hipMalloc((void**)&d_values, filesize ));
    gpuErrchk( hipMalloc((void**)&d_histo, NUMBINS*sizeof(unsigned int) ));

    gpuErrchk( hipMemcpy( d_values, h_values, filesize, hipMemcpyHostToDevice ) );

    computeHistogram( d_values, d_histo, NUMBINS, numElems );  
    gpuErrchk( hipMemcpy( h_histo, d_histo, binsize, hipMemcpyDeviceToHost ));



  }

}
