#include "hip/hip_runtime.h"
#include <iostream>
//#include "hip/hip_runtime.h"
//#include ""
#include <hiprand/hiprand_kernel.h>
#include <ctime>
#include <cstdio>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>

struct saxpy_functor
{
  const float a;

  saxpy_functor(float _a) : a(_a) {}

  __host__ __device__ float operator()(const float& x, const float& y) const
  { 
    return a * x + y;
  }
  
};

__global__ void f_cudaRand(float *d_out){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState state;
    hiprand_init((unsigned long long)clock() + i, 0, 0, &state);

    d_out[i] = hiprand_normal(&state);

}



